/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

    __shared__ float Ashared[TILE_SIZE][TILE_SIZE];
    __shared__ float Bshared[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by * TILE_SIZE + ty;
    int Col = bx * TILE_SIZE + tx;
    float temp = 0.0;

    for (int x = 0; x < (k-1 + TILE_SIZE)/TILE_SIZE ; ++x) 
    {
        if (Row < m && (x*TILE_SIZE+tx)<k )
        {
            Ashared[ty][tx] = A[Row*k + x*TILE_SIZE+tx];
        }
        else
        {
            Ashared[ty][tx] = 0;
        }

        if ((x*TILE_SIZE +ty)< k && Col< n )
        {
            Bshared[ty][tx] = B[(x*TILE_SIZE +ty)*n + Col];
        }
        else
        {
            Bshared[ty][tx] = 0;
        }

        __syncthreads();
        for (int y = 0; y < TILE_SIZE; ++y)
            temp += Ashared[ty][y] * Bshared[y][tx];
        __syncthreads();

    }

    if (Row<m && Col<n)
    {
        C[Row*n+Col] = temp;
    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE

    dim3 dimGrid((n+BLOCK_SIZE-1)/BLOCK_SIZE,(m+BLOCK_SIZE-1)/BLOCK_SIZE,1);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<dimGrid,dimBlock>>>(m, n, k, A, B, C);
}


